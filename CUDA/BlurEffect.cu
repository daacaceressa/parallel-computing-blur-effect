#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sstream>
#include <iomanip>
#include <time.h>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

const int MAX_ROW = 2200, MAX_COL = 4100;
const int NUMBER_OF_BLOCKS = 4;
int kernelSize, numberOfThreads;
Mat image, newImage;
int h_in[ 3 * MAX_ROW * MAX_COL ];
int h_out[ 3 * MAX_ROW * MAX_COL ];
int size;

/*void getNewPixel( int row, int col, int totalRow, int totalCol ){
    //Calculate the value of the pixel [row][col]
    int newColor[3]; newColor[0] = newColor[1] = newColor[2] = 0;
    for( int i = row - kernelSize/2; i <= row + kernelSize/2; ++i ){
        for( int j = col - kernelSize/2; j <= col + kernelSize/2; ++j ){
            for( int k = 0; k < 3; ++k ) {
                newColor[k] +=  h_in[ k ][ (i+totalRow)%totalRow ][ (j+totalCol)%totalCol ];
            }
        }
    }

    //Store it as the variable of a pixel
    for( int k = 0; k < 3; ++k ) {
        h_out[k][row][col] = newColor[k] / (kernelSize*kernelSize);
    }
}*/

/*void blur(){
    newImage = Mat(image.rows, image.cols, CV_8UC3);
    int j;
    omp_set_num_threads( numberOfThreads );
    #pragma omp parallel for
        for( j = 0; j < image.rows; ++j ){
            for( int i = 0; i < image.cols; ++i ){
                getNewPixel( j, i, image.rows, image.cols );
            }
        }
    
}*/

__global__ void blur( int * d_in, int * d_out, int rowsPerThread, int totalRow, int totalCol, int kernelSize ){
    int fr = rowsPerThread * (blockDim.x * blockIdx.x + threadIdx.x);
    int to = fr + rowsPerThread;
    int newColor[3];
    for( int row = fr; row < to && row < totalRow; ++row ){
        for( int col = 0; col < totalCol; ++col ){
            //Calculate the value of the pixel [row][col]
            newColor[0] = newColor[1] = newColor[2] = 0;
            for( int i = row - kernelSize/2; i <= row + kernelSize/2; ++i ){
                for( int j = col - kernelSize/2; j <= col + kernelSize/2; ++j ){
                    for( int k = 0; k < 3; ++k ) {
                        newColor[k] += d_in[ (totalCol*((i+totalRow)%totalRow) + ((j+totalCol)%totalCol))*3+k ];
					
                    }
                }
            }
            //Store it as the variable of a pixel
            for( int k = 0; k < 3; ++k ) {
                d_out[ (totalCol*row + col)*3+k ] = newColor[k] / (kernelSize*kernelSize);
            }
        }
    }
}

void storeImageData(){
    Vec3b currentColor;
    for( int j = 0; j < image.rows; ++j ){
        for( int i = 0; i < image.cols; ++i ){
            currentColor = image.at<Vec3b>(Point( i, j ));
            for( int k = 0; k < 3; ++k ){
                h_in[ (image.cols*j + i)*3+k ] = currentColor[ k ];
            }
        }
    }
}

void saveNewImageData(){
    newImage = Mat(image.rows, image.cols, CV_8UC3);
    for( int j = 0; j < image.rows; ++j ){
        for( int i = 0; i < image.cols; ++i ){
            Vec3b currentPixel;
            for( int k = 0; k < 3; ++k ){
                currentPixel[ k ] = h_out[ (image.cols*j + i)*3+k ];
            }
            newImage.at<Vec3b>(Point( i, j )) = currentPixel;
        }
    }
}

void displayImage( Mat &image ){
    namedWindow("Display Image", WINDOW_AUTOSIZE );
    imshow("Display Image", image);
    waitKey(0);
}

int main(int argc, char** argv )
{
	hipSetDevice(0);
    //start time
    struct timespec start, finish;
    double elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);

    if ( argc != 4 )
    {
        printf("usage: ./script.sh <Image_Path> <Kernel_Size> <Number_Threads>\n");
        return -1;
    }

    //Read original image using path
    image = imread( argv[1], 1 );
    if ( !image.data )
    {
        printf("No image data \n");
        return -1;
    }

    //Read kernel size
    stringstream ss1( argv[ 2 ] );
    ss1 >> kernelSize;
    if( !(kernelSize&1) || kernelSize < 1 ){
        printf( "Kernel size must be an odd positive integer.\n" );
        return -1;
    }

    //Read number of threads
    stringstream ss2( argv[ 3 ] );
    ss2 >> numberOfThreads;
    if( (numberOfThreads < 1) || (numberOfThreads&1) ){
        printf( "Number of threads must be an even positive integer.\n" );
        return -1;
    }

    //cout << "File\t\tKernel\t\tThreads\t\tTime(s)\n";
    printf( "%s\t\t", argv[ 1 ] );
    printf( "%d\t\t\t\t", kernelSize );
    printf( "%d\t\t\t\t", numberOfThreads );


    size = sizeof( int ) * 3 * MAX_COL * MAX_ROW;

    //Declaring pointers
    int * d_in, * d_out;

    //Alloc memory
    hipMalloc( (void **) &d_in, size );
    hipMalloc( (void **) &d_out, size );

    //Initialize variables
    storeImageData();

    //Copy host to device
    hipMemcpy( d_in, &h_in, size, hipMemcpyHostToDevice );

    //Launch kernel
    //blur();
    blur<<< NUMBER_OF_BLOCKS, numberOfThreads/NUMBER_OF_BLOCKS >>>( d_in, d_out, (image.rows + numberOfThreads - 1)/numberOfThreads, image.rows, image.cols, kernelSize );
    
    //Copy device to host
    hipMemcpy( &h_out, d_out, size, hipMemcpyDeviceToHost );

    //Free memory
    hipFree( d_in );
    hipFree( d_out );

    //Create newImage with the matrix out
    saveNewImageData();



    //calculate and print elapsed time
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    printf( "%.4f\n", elapsed );

    //Display blurred image
    //displayImage( newImage );

    return 0;
}
